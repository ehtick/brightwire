#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

#define BLOCKSIZE 32
#define N BLOCKSIZE*BLOCKSIZE
#define NEG_INF __int_as_float(0xff800000)
#define POS_INF __int_as_float(0x7f800000)

typedef unsigned int uint;

extern "C"
{
    __global__ void IsFinite(const float* __restrict a, float* __restrict b, uint size, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = isfinite(a[index * ai]) ? 0 : 1;
        }
	}

    __global__ void Scale(float* __restrict a, uint size, float scale, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index * ai] *= scale;
        }
	}

	__global__ void PointwiseMultiply(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index * bi] *= a[index * ai];
        }
	}

	__global__ void PointwiseDivide(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index * bi] = a[index * ai] / b[index * bi];
        }
	}

	__global__ void Sqrt(const float* __restrict a, float* __restrict b, uint size, float valueAdjustment, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index * bi] = sqrt(a[index * ai] + valueAdjustment);
        }
	}

	__global__ void AddInPlace(float* __restrict a, const float* __restrict b, uint size, float coefficient1, float coefficient2, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index * ai] = (a[index * ai] * coefficient1) + (b[index * bi] * coefficient2);
        }
	}

	__global__ void SubtractInPlace(float* __restrict a, const float* __restrict b, uint size, float coefficient1, float coefficient2, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index * ai] = (a[index * ai] * coefficient1) - (b[index * bi] * coefficient2);
        }
	}

	__global__ void AddToEachRow(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[j];
            }
        }
	}

	__global__ void AddToEachColumn(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[i];
            }
        }
	}

    __global__ void MultiplyByEachRow(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] *= b[j];
            }
        }
	}

	__global__ void MultiplyByEachColumn(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] *= b[i];
            }
        }
	}

	__global__ void TanH(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index * bi] = tanh(a[index * ai]);
        }
	}

	__global__ void TanHDerivative(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float ta = tanh(a[index * ai]);
            b[index * bi] = 1.0f - ta * ta;
        }
	}

	__global__ void Sigmoid(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index * bi] = 1.0f / (1.0f + exp(-1.0f * a[index * ai]));
        }
	}

	__global__ void SigmoidDerivative(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float sigmoid = 1.0f / (1.0f + exp(-1.0f * a[index * ai]));
			b[index * bi] = sigmoid * (1.0f - sigmoid);
        }
	}

	__global__ void RELU(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index * ai];
			b[index * bi] = (val <= 0) ? 0 : val;
        }
	}

	__global__ void RELUDerivative(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index * ai];
			b[index * bi] = (val <= 0) ? 0 : 1;
        }
	}

	__global__ void LeakyRELU(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index * ai];
			b[index * bi] = (val <= 0) ? 0.01f*val : val;
        }
	}

	__global__ void LeakyRELUDerivative(const float* __restrict a, float* __restrict b, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index * ai];
			b[index * bi] = (val <= 0) ? 0.01f : 1;
        }
	}

	__global__ void Reverse(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[size - index - 1] = a[index];
        }
	}

	__global__ void SumRows(const float* __restrict a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                atomicAdd(b + i, a[j * rows + i]);
            }
        }
	}

	__global__ void SumColumns(const float* __restrict a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                atomicAdd(b + j, a[j * rows + i]);
            }
        }
	}

	__global__ void MemSet(float* a, float val, uint count, uint offset, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
			a[offset + (index * ai)] = val;
        }
	}

    __global__ void MemCpy(float* __restrict a, float* __restrict b, uint count, uint offsetA, uint offsetB, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
			a[offsetA + (index * ai)] = b[offsetB + (index * bi)];
        }
	}

	__global__ void FindMinAndMax(const float* __restrict a, uint count, float* __restrict minBlock, float* __restrict maxBlock, uint ai)
	{
		uint tidX = threadIdx.x;
		uint blockX = blockIdx.x;
		uint index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[N];
		block[tidX] = (index < count) ? a[index * ai] : 0;
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float min = FLT_MAX, max = FLT_MIN;
			uint maxIndex = N;
			if (count - index < N)
				maxIndex = count - index;
			for (uint i = 0; i < maxIndex; i++) {
				float val = block[i];
				if (i == 0 || val > max)
					max = val;
				if (i == 0 || val < min)
					min = val;
			}
			minBlock[blockX] = min;
			maxBlock[blockX] = max;
		}
	}

	__global__ void FindStdDev(const float* __restrict a, uint count, float mean, float* __restrict stdDev, uint ai)
	{
		uint tidX = threadIdx.x;
		uint blockX = blockIdx.x;
		uint index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[N];
		if (index < count)
			block[tidX] = a[index * ai];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			uint maxIndex = N;
			if (count - blockX * N < N)
				maxIndex = count - blockX * N;
			for (uint i = 0; i < maxIndex; i++) {
                float val = block[i] - mean;
				total += val * val;
			}
			stdDev[blockX] = total;
		}
	}

	__global__ void Constrain(float* __restrict a, uint count, float min, float max, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            uint ind = index * ai;
            float val = a[ind];
			if (val < min || val == NEG_INF)
				a[ind] = min;
			if (val > max || val == POS_INF)
				a[ind] = max;
            if(isnan(val))
                a[ind] = 0;
        }
	}

    __global__ void RoundInPlace(float* __restrict a, uint count, float lower, float upper, float mid, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            uint ind = index * ai;
            float val = a[ind * ai];
			if (val >= mid)
				a[ind * ai] = upper;
			else
				a[ind * ai] = lower;
        }
	}

	__global__ void Pow(const float* __restrict a, float* __restrict b, uint count, float power, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
			b[index * bi] = pow(a[index * ai], power);
        }
	}

	__global__ void Diagonal(const float* __restrict a, float* __restrict b, uint rows, uint columns, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < rows && index < columns; index += blockDim.x * gridDim.x) {
            b[index * bi] = a[index * ai * rows + index];
        }
	}

	__global__ void L1Regularisation(float* __restrict a, uint count, float coefficient, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index * ai];
			a[index * ai] = val - ((val > 0 ? 1 : val < 0 ? -1 : 0) * coefficient);
        }
	}

	__global__ void PointwiseDivideRows(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                uint index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[i];
            }
        }
	}

	__global__ void PointwiseDivideColumns(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                uint index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[j];
            }
        }
	}

	__global__ void SplitRows(const float* __restrict a, float* __restrict b, float* __restrict c, uint rows, uint columns, uint position)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                uint index = j * rows + i;
			    float val = a[index];
			    if(j >= position) {
				    uint diff = j - position;
				    c[diff * rows + i] = val;
			    }else
				    b[index] = val;
            }
        }
	}

	__global__ void SplitColumns(const float* __restrict a, float* __restrict b, float* __restrict c, uint rows, uint columns, uint position)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j * rows + i];
			    if(i >= position) {
				    uint diff = i - position;
				    c[j * (rows-position) + diff] = val;
			    }else
				    b[j * position + i] = val;
            }
        }
	}

	__global__ void ConcatColumns(const float* __restrict a, const float* __restrict b, float* __restrict c, uint rows, uint columns, uint topRowCount, uint bottomRowCount)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(i >= topRowCount)
				    val = b[j * bottomRowCount + i - topRowCount];
			    else
				    val = a[j * topRowCount + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void ConcatRows(const float* __restrict a, const float* __restrict b, float* __restrict c, uint rows, uint columns, uint leftColumnCount)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(j >= leftColumnCount)
				    val = b[(j-leftColumnCount) * rows + i];
			    else
				    val = a[j * rows + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void EuclideanDistance(const float* __restrict a, const float* __restrict b, float* __restrict c, uint count, uint ai, uint bi, uint ci)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index * ai] - b[index * bi];
            c[index * ci] = val * val;
        }
	}

	/*__global__ void MultiEuclideanDistance(const float* __restrict a, const float* __restrict* b, float* __restrict c, uint size, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
                float val3 = val1 - val2;
			    c[j * size + i] = val3 * val3;
            }
        }
	}*/

	__global__ void ManhattanDistance(const float* __restrict a, const float* __restrict b, float* __restrict c, uint count, uint ai, uint bi, uint ci)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            c[index * ci] = abs(a[index * ai] - b[index * bi]);
        }
	}

	/*__global__ void MultiManhattanDistance(const float* __restrict a, const float* __restrict* b, float* __restrict c, uint size, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
			    c[j * size + i] = abs(val1 - val2);
            }
        }
	}*/

	__global__ void CosineDistance(const float* __restrict a, const float* __restrict b, float* __restrict aa, float* __restrict ab, float* __restrict bb, uint count, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
			float left = a[index * ai];
			float right = b[index * bi];
			atomicAdd(aa, left * left);
			atomicAdd(ab, left * right);
			atomicAdd(bb, right * right);
        }
	}

	__global__ void Abs(const float* __restrict a, float* __restrict b, uint count, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index * bi] = abs(a[index * ai]);
        }
	}

	__global__ void Log(const float* __restrict a, float* __restrict b, uint count, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index * bi] = log(a[index * ai]);
        }
	}

    __global__ void Exp(const float* __restrict a, float* __restrict b, uint count, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index * bi] = exp(a[index * ai]);
        }
	}

	__global__ void Normalise(float* __restrict a, uint count, float min, float range, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            a[index * ai] = (a[index * ai] - min) / range;
        }
	}

	__global__ void SoftmaxVector(const float* __restrict a, float* __restrict b, uint count, float max, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = exp(a[index * ai] - max);
        }
	}

	__global__ void VectorAddInPlace(float* __restrict a, uint size, float scalar, uint ai)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index * ai] += scalar;
        }
	}

	__global__ void VectorCopyRandom(const float* __restrict a, float* __restrict b, uint* __restrict c, uint size, uint ai, uint bi)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
			b[index * bi] += a[c[index] * ai];
        }
	}

	__global__ void CopyToMatrixRows(const float* __restrict* a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[i][j];
			    b[j * rows + i] = val;
            }
        }
	}

    __global__ void CopyToMatrixColumns(const float* __restrict* a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j][i];
                //printf("i:%i(%i) j:%i(%i)\n", i, rows, j, columns);
			    b[j * rows + i] = val;
            }
        }
	}

	__global__ void TensorAddPadding(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        uint rows, 
        uint columns, 
        uint depth, 
        uint count, 
        uint outputRows, 
        uint outputColumns, 
        uint padding
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint i = index % outputRows;
            uint index2 = index / outputRows;

            uint j = index2 % outputColumns;
            uint index3 = index2 / outputColumns;

            uint k = index3 % depth;
            uint z = index3 / depth;

            float val = 0;
            if(i >= padding && i < (outputRows - padding) && j >= padding && j < (outputColumns - padding)) {
                const float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                uint aIndex = (j-padding) * rows + (i-padding);
                val = inputPtr[aIndex];

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, val
                );*/
            }

            float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
            outputPtr[j * outputRows + i] = val;
        }
	}

	__global__ void TensorRemovePadding(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        uint rows, 
        uint columns, 
        uint depth, 
        uint count, 
        uint outputRows, 
        uint outputColumns, 
        uint padding
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint i = index % rows;
            uint index2 = index / rows;

            uint j = index2 % columns;
            if(i >= padding && i < (rows-padding) && j >= padding && j < (columns-padding)) {
                uint index3 = index2 / columns;

                uint k = index3 % depth;
                uint z = index3 / depth;

                const float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                uint aIndex = j * rows + i;
                float val = inputPtr[aIndex];

                float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
                uint bIndex = (j-padding) * outputRows + (i-padding);
                outputPtr[bIndex] = val;

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i bi:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, bIndex,
                    val
                );*/
            }
        }
	}

    __global__ void TensorIm2Col(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        const float* __restrict cx, 
        const float* __restrict cy, 
        uint rows,
        uint columns,
        uint depth,
        uint count,
        uint outputRows,
        uint outputColumns,
        uint convolutionCount, 
        uint filterWidth, 
        uint filterHeight,
        uint xStride,
		uint yStride
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint x = index % filterWidth;
            uint index2 = index / filterWidth;

            uint y = index2 % filterHeight;
            uint index3 = index2 / filterHeight;

            uint k = index3 % depth;
            uint index4 = index3 / depth;

            uint ci = index4 % convolutionCount;
            uint i = index4 / convolutionCount;

            uint offsetX = cx[ci];
            uint offsetY = cy[ci];

            /*printf("index:%i, i:%i(%i), ci:%i(%i), k:%i(%i), x:%i(%i), y:%i(%i), cx:%i=%i, cy:%i=%i\n", index,
                i, count,
                ci, convolutionCount,
                k, depth,
                x, filterWidth,
                y, filterHeight,
                offsetX, (uint)cx[ci],
                offsetY, (uint)cy[ci]
            );*/

            uint filterOffset = k * filterWidth * filterHeight;
            uint filterIndex = filterOffset + (x * filterHeight + y);

            float* outputPtr = b + (outputRows * outputColumns * i);
            const float* inputPtr = a + (rows * columns * depth * i) + (rows * columns * k);
            outputPtr[filterIndex * outputRows + ci] = inputPtr[(offsetX + x) * rows + (offsetY + y)];
        }
    }

    __global__ void TensorReverseIm2Col(
        uint size, 
        const float* __restrict a, 
        const float* __restrict filters, 
        float* __restrict b, 
        const float* __restrict cx, 
        const float* __restrict cy, 
        uint rows, 
        uint columns, 
        uint depth, 
        uint count,
        uint convolutionCount,  
        uint filterWidth, 
        uint filterHeight, 
        uint xStride,
		uint yStride,
        uint outputRows,
        uint outputColumns,
        uint outputDepth
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint z = index % outputDepth;
            uint index2 = index / outputDepth;

            uint x = index2 % filterWidth;
            uint index3 = index2 / filterWidth;

            uint y = index3 % filterHeight;
            uint index4 = index3 / filterHeight;

            uint ci = index4 % convolutionCount;
            uint index5 = index4 / convolutionCount;

            uint k = index5 % depth;
            uint i = index5 / depth;

            uint offsetX = cx[ci];
            uint offsetY = cy[ci];

            /*printf("index:%i di:%i(%i) ci:%i(%i) k:%i(%i) x:%i(%i) y:%i(%i) z:%i(%i) cx:%i cy:%i\n", index,
                i, count,
                ci, convolutionCount, 
                k, depth, 
                x, filterWidth, 
                y, filterHeight, 
                z, outputDepth, 
                offsetX, offsetY
            );*/

            const float* slice = a + (i * rows * columns * depth) + (k * rows * columns);
            const float* filter = filters + (k * outputDepth * filterWidth * filterHeight) + (z * filterWidth * filterHeight);
            float* output = b + (i * outputRows * outputColumns * outputDepth) + (z * outputRows * outputColumns);

            uint errorX = offsetX / xStride;
            uint errorY = offsetY / yStride;
            if(errorX < columns && errorY < rows) {
                float error = slice[errorX * rows + errorY];

                uint filterIndex = (filterWidth-x-1) * filterHeight + (filterHeight-y-1);
                uint outputIndex = (offsetX+x) * outputRows + (offsetY+y);
                float val = filter[filterIndex] * error;

                atomicAdd(output + outputIndex, val);
            }
        }
    }

	__global__ void SoftmaxDerivative(const float* __restrict a, float* __restrict b, uint size, uint ai)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < size; j += blockDim.y * gridDim.y) {
                uint index = j * size + i;
			    if(i == j)
				    b[index] = a[i * ai] * (1 - a[i * ai]);
			    else
				    b[index] = -a[i * ai] * a[j * ai];
            }
        }
	}

	__global__ void RotateInPlace(float* __restrict a, uint size, uint blockCount, uint blockSize)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint blockIndex = index / blockSize;
			uint blockOffset = index % blockSize;
            uint index1 = blockIndex * blockSize + blockSize - blockOffset - 1;
			uint index2 = blockIndex * blockSize + blockOffset; 
			float temp = a[index1];
			a[index1] = a[index2];
			a[index2] = temp;
        }
	}

	__global__ void TensorMaxPool(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        float* __restrict indexOffset,
        const float* __restrict cx, 
        const float* __restrict cy,
        uint convolutionCount,
        uint rows, 
        uint columns, 
        uint depth, 
        uint count, 
        uint outputRows, 
        uint outputColumns, 
        uint filterWidth, 
        uint filterHeight, 
        uint xStride,
		uint yStride,
        uint saveIndices
    ) {
		for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint ci = index % convolutionCount;
            uint index2 = index / convolutionCount;

            uint k = index2 % depth;
            uint z = index2 / depth;

            uint aX = cx[ci];
			uint aY = cy[ci];
            uint bX = aX / xStride;
            uint bY = aY / yStride;

            /*printf("index:%i k:%i(%i) z:%i(%i) ax:%i ay:%i bx:%i by:%i\n", index,
                k, depth, 
                z, count,
                aX, aY,
                bX, bY
            );*/

            uint targetOffset = (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            const float* source = a + (z * rows * columns * depth) + (k * rows * columns);
            float* target = b + targetOffset;

            float maxVal = 0;
	        int bestOffset = -1;
	        uint offset = 0;
	                
	        for (uint x = 0; x < filterWidth; x++) {
		        for (uint y = 0; y < filterHeight; y++) {
			        float val = source[(aX + x) * rows + (aY + y)];
                    bool isGreater = (bestOffset < 0 || val > maxVal);
			        if (isGreater) {
				        bestOffset = offset;
				        maxVal = val;
			        }
                    //printf("index:%i, x:%i, y:%i val:%f max:%f offset:%i is-greater:%i\n", index, x, y, val, maxVal, bestOffset, isGreater ? 1 : 0);
					++offset;
		        }
	        }

            //printf("\tindex:%i i:%i j:%i val:%f\n", index, i, j, maxVal);
            if(saveIndices) {
                float* indices = indexOffset + targetOffset;
                indices[bX * outputRows + bY] = bestOffset;
            }
            target[bX * outputRows + bY] = maxVal;
        }
	}

	__global__ void TensorReverseMaxPool(
        uint size, 
        const float* __restrict a,
        const float* __restrict indices,
        float* __restrict b, 
        uint rows,
        uint columns,
        uint depth,
        uint count,
        uint outputRows,
        uint outputColumns,
        uint filterWidth,
        uint filterHeight,
        uint xStride,
		uint yStride
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint i = index % rows;
            uint index2 = index / rows;

            uint j = index2 % columns;
            uint index3 = index2 / columns;

            uint k = index3 % depth;
            uint z = index3 / depth;

            uint sourceOffset = (z * rows * columns * depth) + (k * rows * columns);
            const float* source = a + sourceOffset;
            const float* indexPtr = indices + sourceOffset;
            float* target = b + (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            uint sourceIndex = j * rows + i;
            float val = source[sourceIndex];
            int offset = indexPtr[sourceIndex];

            if(offset < 0)
                offset = 0;

            uint targetX = j * xStride + (offset / filterHeight);
            uint targetY = i * yStride + (offset % filterHeight);

            /*printf("index:%i s:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) val:%f offset:%i tx:%i ty:%i\n", 
                index, xStride, yStride
                i, outputRows,
                j, outputColumns, 
                k, depth, 
                z, count,
                val, offset,
                targetX, targetY
            );*/

            target[targetX * outputRows + targetY] = val;
        }
	}

    __global__ void CalculateMultiDistances(
        const float** __restrict a,
        const float** __restrict b,
        float* __restrict c,
        uint rows,
        uint columns,
        uint size,
        uint distanceMetric
    ) {
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                for (uint k = blockDim.z * blockIdx.z + threadIdx.z; k < rows; k += blockDim.z * gridDim.z) {
                    float aVal = a[j][i];
                    float bVal = b[k][i];
                    float output = 0;

                    if(distanceMetric == 0) { // euclidean
                        float diff = aVal - bVal;
                        output = diff * diff;
                    }else if(distanceMetric == 2) { // manhattan
                        output = abs(aVal - bVal);
                    }
                    float* outputPtr = c + (j * rows + k);
                    atomicAdd(outputPtr, output);
                }
            }
        }
	}

    __global__ void CalculateDistances(
        const float* __restrict a,
        const float** __restrict b,
        float* __restrict c,
        uint numVectors,
        uint size,
        uint distanceMetric
    ) {
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < numVectors; j += blockDim.y * gridDim.y) {
                float aVal = a[i];
                float bVal = b[j][i];
                float output = 0;

                if(distanceMetric == 0) { // euclidean
                    float diff = aVal - bVal;
                    output = diff * diff;
                }else if(distanceMetric == 2) { // manhattan
                    output = abs(aVal - bVal);
                }
                atomicAdd(c + j, output);
            }
        }
	}

	__global__ void CosineMultiDistance(
		const float** __restrict a, 
		const float** __restrict b, 
		float* __restrict aa, 
		float* __restrict ab, 
		float* __restrict bb, 
		uint rows,
        uint columns,
        uint size
	) {
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                for (uint k = blockDim.z * blockIdx.z + threadIdx.z; k < rows; k += blockDim.z * gridDim.z) {
					float aVal = a[j][i];
					float bVal = b[k][i];
					uint offset = j * rows + k;
					atomicAdd(aa + offset, aVal * aVal);
					atomicAdd(ab + offset, aVal * bVal);
					atomicAdd(bb + offset, bVal * bVal);
				}
            }
        }
	}

    __global__ void CosineDistances(
		const float* __restrict a, 
		const float** __restrict b, 
		float* __restrict aa, 
		float* __restrict ab, 
		float* __restrict bb, 
		uint numVectors,
        uint size
	) {
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < numVectors; j += blockDim.y * gridDim.y) {
				float aVal = a[i];
				float bVal = b[j][i];
				atomicAdd(aa + j, aVal * aVal);
				atomicAdd(ab + j, aVal * bVal);
				atomicAdd(bb + j, bVal * bVal);
            }
        }
	}

    __global__ void SumValues(const float* __restrict a, uint count, float* __restrict sum, uint ai)
	{
		uint tidX = threadIdx.x;
		uint blockX = blockIdx.x;
		uint index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[N];
		if (index < count)
			block[tidX] = a[index * ai];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			uint maxIndex = N;
			if (count - blockX * N < N)
				maxIndex = count - blockX * N;
			for (uint i = 0; i < maxIndex; i++) {
				total += block[i];
			}
			sum[blockX] = total;
		}
	}
}