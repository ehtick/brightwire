#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

#define BLOCKSIZE 16
#define BLOCKSIZE2 BLOCKSIZE*BLOCKSIZE

typedef unsigned int uint;

extern "C"
{
    __global__ void IsFinite(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = isfinite(a[index]) ? 0 : 1;
        }
	}

	__global__ void PointwiseMultiply(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] *= a[index];
        }
	}

	__global__ void PointwiseDivide(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = a[index] / b[index];
        }
	}

	__global__ void Sqrt(const float* __restrict a, float* __restrict b, uint size, float valueAdjustment)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = sqrt(a[index] + valueAdjustment);
        }
	}

	__global__ void AddInPlace(float* __restrict a, const float* __restrict b, uint size, float coefficient1, float coefficient2)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] * coefficient1) + (b[index] * coefficient2);
        }
	}

	__global__ void SubtractInPlace(float* __restrict a, const float* __restrict b, uint size, float coefficient1, float coefficient2)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] * coefficient1) - (b[index] * coefficient2);
        }
	}

	__global__ void AddToEachRow(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[j];
            }
        }
	}

	__global__ void AddToEachColumn(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[i];
            }
        }
	}

	__global__ void TanH(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = tanh(a[index]);
        }
	}

	__global__ void TanHDerivative(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = 1.0f - pow(tanh(a[index]), 2);
        }
	}

	__global__ void Sigmoid(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = 1.0f / (1.0f + exp(-1.0f * a[index]));
        }
	}

	__global__ void SigmoidDerivative(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float sigmoid = 1.0f / (1.0f + exp(-1.0f * a[index]));
			b[index] = sigmoid * (1.0f - sigmoid);
        }
	}

	__global__ void RELU(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0 : val;
        }
	}

	__global__ void RELUDerivative(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0 : 1;
        }
	}

	__global__ void LeakyRELU(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0.01f*val : val;
        }
	}

	__global__ void LeakyRELUDerivative(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0.01f : 1;
        }
	}

	__global__ void Reverse(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[size - index - 1] = a[index];
        }
	}

	__global__ void SumRows(const float* __restrict a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                atomicAdd(b + i, a[j * rows + i]);
            }
        }
	}

	__global__ void SumColumns(const float* __restrict a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                atomicAdd(b + j, a[j * rows + i]);
            }
        }
	}

	__global__ void MemClear(float* data, uint count, uint srcOffset, uint srcIncrement)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            uint arrayIndex = srcOffset + (index * srcIncrement);
			data[arrayIndex] = 0.0f;
        }
	}

	__global__ void FindMinAndMax(const float* __restrict data, uint count, float* __restrict minBlock, float* __restrict maxBlock)
	{
		uint tidX = threadIdx.x;
		uint blockX = blockIdx.x;
		uint index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		block[tidX] = (index < count) ? data[index] : 0;
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float min = FLT_MAX, max = FLT_MIN;
			uint maxIndex = BLOCKSIZE2;
			if (count - index < BLOCKSIZE2)
				maxIndex = count - index;
			for (uint i = 0; i < maxIndex; i++) {
				float val = block[i];
				if (i == 0 || val > max)
					max = val;
				if (i == 0 || val < min)
					min = val;
			}
			minBlock[blockX] = min;
			maxBlock[blockX] = max;
		}
	}

	__global__ void FindSum(const float* __restrict data, uint count, float* __restrict sum)
	{
		uint tidX = threadIdx.x;
		uint blockX = blockIdx.x;
		uint index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			uint maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (uint i = 0; i < maxIndex; i++) {
				total += block[i];
			}
			sum[blockX] = total;
		}
	}

	__global__ void FindStdDev(const float* __restrict data, uint count, float mean, float* __restrict stdDev)
	{
		uint tidX = threadIdx.x;
		uint blockX = blockIdx.x;
		uint index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			uint maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (uint i = 0; i < maxIndex; i++) {
				total += pow(block[i] - mean, 2);
			}
			stdDev[blockX] = total;
		}
	}

	__global__ void Constrain(float* __restrict data, uint count, float min, float max)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = data[index];
			if (val < min)
				data[index] = min;
			if (val > max)
				data[index] = max;
        }
	}

	__global__ void Pow(const float* __restrict a, float* __restrict b, uint count, float power)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = pow(val, power);
        }
	}

	__global__ void Diagonal(const float* __restrict a, float* __restrict b, uint rows, uint columns)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < rows && index < columns; index += blockDim.x * gridDim.x) {
            b[index] = a[index * rows + index];
        }
	}

	__global__ void L1Regularisation(float* __restrict a, uint count, float coefficient)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index];
			a[index] = val - ((val > 0 ? 1 : val < 0 ? -1 : 0) * coefficient);
        }
	}

	__global__ void PointwiseDivideRows(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                uint index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[i];
            }
        }
	}

	__global__ void PointwiseDivideColumns(float* __restrict a, const float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                uint index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[j];
            }
        }
	}

	__global__ void SplitRows(const float* __restrict a, float* __restrict b, float* __restrict c, uint rows, uint columns, uint position)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                uint index = j * rows + i;
			    float val = a[index];
			    if(j >= position) {
				    uint diff = j - position;
				    c[diff * rows + i] = val;
			    }else
				    b[index] = val;
            }
        }
	}

	__global__ void SplitColumns(const float* __restrict a, float* __restrict b, float* __restrict c, uint rows, uint columns, uint position)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j * rows + i];
			    if(i >= position) {
				    uint diff = i - position;
				    c[j * (rows-position) + diff] = val;
			    }else
				    b[j * position + i] = val;
            }
        }
	}

	__global__ void ConcatColumns(const float* __restrict a, const float* __restrict b, float* __restrict c, uint rows, uint columns, uint topRowCount, uint bottomRowCount)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(i >= topRowCount)
				    val = b[j * bottomRowCount + i - topRowCount];
			    else
				    val = a[j * topRowCount + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void ConcatRows(const float* __restrict a, const float* __restrict b, float* __restrict c, uint rows, uint columns, uint leftColumnCount)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(j >= leftColumnCount)
				    val = b[(j-leftColumnCount) * rows + i];
			    else
				    val = a[j * rows + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void EuclideanDistance(const float* __restrict a, const float* __restrict b, float* __restrict c, uint count)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            c[index] = pow(a[index] - b[index], 2);
        }
	}

	/*__global__ void MultiEuclideanDistance(const float* __restrict a, const float* __restrict* b, float* __restrict c, uint size, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
			    c[j * size + i] = pow(val1 - val2, 2);
            }
        }
	}*/

	__global__ void ManhattanDistance(const float* __restrict a, const float* __restrict b, float* __restrict c, uint count)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            c[index] = abs(a[index] - b[index]);
        }
	}

	/*__global__ void MultiManhattanDistance(const float* __restrict a, const float* __restrict* b, float* __restrict c, uint size, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
			    c[j * size + i] = abs(val1 - val2);
            }
        }
	}*/

	__global__ void CosineDistance(const float* __restrict a, const float* __restrict b, float* __restrict aa, float* __restrict ab, float* __restrict bb, uint count)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
			float left = a[index];
			float right = b[index];
			atomicAdd(aa, left * left);
			atomicAdd(ab, left * right);
			atomicAdd(bb, right * right);
        }
	}

	__global__ void Abs(const float* __restrict a, float* __restrict b, uint count)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = abs(a[index]);
        }
	}

	__global__ void Log(const float* __restrict a, float* __restrict b, uint count)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = log(a[index]);
        }
	}

	__global__ void Normalise(float* __restrict a, uint count, float min, float range)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] - min) / range;
        }
	}

	__global__ void SoftmaxVector(const float* __restrict a, float* __restrict b, uint count, float max)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = exp(a[index] - max);
        }
	}

	__global__ void VectorAdd(float* __restrict a, uint size, float scalar)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] += scalar;
        }
	}

	__global__ void VectorCopyRandom(const float* __restrict a, float* __restrict b, uint* __restrict c, uint size)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
			b[index] += a[c[index]];
        }
	}

	__global__ void CopyToMatrixRows(const float* __restrict* a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[i][j];
			    b[j * rows + i] = val;
            }
        }
	}

    __global__ void CopyToMatrixColumns(const float* __restrict* a, float* __restrict b, uint rows, uint columns)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j][i];
                //printf("i:%i(%i) j:%i(%i)\n", i, rows, j, columns);
			    b[j * rows + i] = val;
            }
        }
	}

	__global__ void TensorAddPadding(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        uint rows, 
        uint columns, 
        uint depth, 
        uint count, 
        uint outputRows, 
        uint outputColumns, 
        uint padding
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint i = index % outputRows;
            uint index2 = index / outputRows;

            uint j = index2 % outputColumns;
            uint index3 = index2 / outputColumns;

            uint k = index3 % depth;
            uint z = index3 / depth;

            float val = 0;
            if(i >= padding && i < (outputRows - padding) && j >= padding && j < (outputColumns - padding)) {
                const float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                uint aIndex = (j-padding) * rows + (i-padding);
                val = inputPtr[aIndex];

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, val
                );*/
            }

            float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
            outputPtr[j * outputRows + i] = val;
        }
	}

	__global__ void TensorRemovePadding(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        uint rows, 
        uint columns, 
        uint depth, 
        uint count, 
        uint outputRows, 
        uint outputColumns, 
        uint padding
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint i = index % rows;
            uint index2 = index / rows;

            uint j = index2 % columns;
            if(i >= padding && i < (rows-padding) && j >= padding && j < (columns-padding)) {
                uint index3 = index2 / columns;

                uint k = index3 % depth;
                uint z = index3 / depth;

                const float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                uint aIndex = j * rows + i;
                float val = inputPtr[aIndex];

                float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
                uint bIndex = (j-padding) * outputRows + (i-padding);
                outputPtr[bIndex] = val;

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i bi:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, bIndex,
                    val
                );*/
            }
        }
	}

    __global__ void TensorIm2Col(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        const float* __restrict cx, 
        const float* __restrict cy, 
        uint rows,
        uint columns,
        uint depth,
        uint count,
        uint outputRows,
        uint outputColumns,
        uint convolutionCount, 
        uint filterWidth, 
        uint filterHeight,
        uint xStride,
		uint yStride
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint x = index % filterWidth;
            uint index2 = index / filterWidth;

            uint y = index2 % filterHeight;
            uint index3 = index2 / filterHeight;

            uint k = index3 % depth;
            uint index4 = index3 / depth;

            uint ci = index4 % convolutionCount;
            uint i = index4 / convolutionCount;

            uint offsetX = cx[ci];
            uint offsetY = cy[ci];

            /*printf("index:%i, i:%i(%i), ci:%i(%i), k:%i(%i), x:%i(%i), y:%i(%i), cx:%i=%i, cy:%i=%i\n", index,
                i, count,
                ci, convolutionCount,
                k, depth,
                x, filterWidth,
                y, filterHeight,
                offsetX, (uint)cx[ci],
                offsetY, (uint)cy[ci]
            );*/

            uint filterOffset = k * filterWidth * filterHeight;
            uint filterIndex = filterOffset + (x * filterHeight + y);

            float* outputPtr = b + (outputRows * outputColumns * i);
            const float* inputPtr = a + (rows * columns * depth * i) + (rows * columns * k);
            outputPtr[filterIndex * outputRows + ci] = inputPtr[(offsetX + x) * rows + (offsetY + y)];
        }
    }

    __global__ void TensorReverseIm2Col(
        uint size, 
        const float* __restrict a, 
        const float* __restrict filters, 
        float* __restrict b, 
        const float* __restrict cx, 
        const float* __restrict cy, 
        uint rows, 
        uint columns, 
        uint depth, 
        uint count,
        uint convolutionCount,  
        uint filterWidth, 
        uint filterHeight, 
        uint xStride,
		uint yStride,
        uint outputRows,
        uint outputColumns,
        uint outputDepth
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint z = index % outputDepth;
            uint index2 = index / outputDepth;

            uint x = index2 % filterWidth;
            uint index3 = index2 / filterWidth;

            uint y = index3 % filterHeight;
            uint index4 = index3 / filterHeight;

            uint ci = index4 % convolutionCount;
            uint index5 = index4 / convolutionCount;

            uint k = index5 % depth;
            uint i = index5 / depth;

            uint offsetX = cx[ci];
            uint offsetY = cy[ci];

            /*printf("index:%i di:%i(%i) ci:%i(%i) k:%i(%i) x:%i(%i) y:%i(%i) z:%i(%i) cx:%i cy:%i\n", index,
                i, count,
                ci, convolutionCount, 
                k, depth, 
                x, filterWidth, 
                y, filterHeight, 
                z, outputDepth, 
                offsetX, offsetY
            );*/

            const float* slice = a + (i * rows * columns * depth) + (k * rows * columns);
            const float* filter = filters + (k * outputDepth * filterWidth * filterHeight) + (z * filterWidth * filterHeight);
            float* output = b + (i * outputRows * outputColumns * outputDepth) + (z * outputRows * outputColumns);

            uint errorX = offsetX / xStride;
            uint errorY = offsetY / yStride;
            if(errorX < columns && errorY < rows) {
                float error = slice[errorX * rows + errorY];

                uint filterIndex = (filterWidth-x-1) * filterHeight + (filterHeight-y-1);
                uint outputIndex = (offsetX+x) * outputRows + (offsetY+y);
                float val = filter[filterIndex] * error;

                atomicAdd(output + outputIndex, val);
            }
        }
    }

	__global__ void SoftmaxDerivative(const float* __restrict a, float* __restrict b, uint size)
	{
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < size; j += blockDim.y * gridDim.y) {
                uint index = j * size + i;
			    if(i == j)
				    b[index] = a[i] * (1 - a[i]);
			    else
				    b[index] = -a[i] * a[j];
            }
        }
	}

	__global__ void RotateInPlace(float* __restrict a, uint size, uint blockCount, uint blockSize)
	{
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint blockIndex = index / blockSize;
			uint blockOffset = index % blockSize;
            uint index1 = blockIndex * blockSize + blockSize - blockOffset - 1;
			uint index2 = blockIndex * blockSize + blockOffset; 
			float temp = a[index1];
			a[index1] = a[index2];
			a[index2] = temp;
        }
	}

	__global__ void TensorMaxPool(
        uint size, 
        const float* __restrict a, 
        float* __restrict b, 
        float* __restrict indexOffset,
        const float* __restrict cx, 
        const float* __restrict cy,
        uint convolutionCount,
        uint rows, 
        uint columns, 
        uint depth, 
        uint count, 
        uint outputRows, 
        uint outputColumns, 
        uint filterWidth, 
        uint filterHeight, 
        uint xStride,
		uint yStride,
        uint saveIndices
    ) {
		for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint ci = index % convolutionCount;
            uint index2 = index / convolutionCount;

            uint k = index2 % depth;
            uint z = index2 / depth;

            uint aX = cx[ci];
			uint aY = cy[ci];
            uint bX = aX / xStride;
            uint bY = aY / yStride;

            /*printf("index:%i k:%i(%i) z:%i(%i) ax:%i ay:%i bx:%i by:%i\n", index,
                k, depth, 
                z, count,
                aX, aY,
                bX, bY
            );*/

            uint targetOffset = (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            const float* source = a + (z * rows * columns * depth) + (k * rows * columns);
            float* target = b + targetOffset;

            float maxVal = 0;
	        uint bestOffset = -1;
	        uint offset = 0;
	                
	        for (uint x = 0; x < filterWidth; x++) {
		        for (uint y = 0; y < filterHeight; y++) {
			        float val = source[(aX + x) * rows + (aY + y)];
                    bool isGreater = (bestOffset < 0 || val > maxVal);
			        if (isGreater) {
				        bestOffset = offset;
				        maxVal = val;
			        }
                    //printf("index:%i, x:%i, y:%i val:%f max:%f offset:%i is-greater:%i\n", index, x, y, val, maxVal, bestOffset, isGreater ? 1 : 0);
					++offset;
		        }
	        }

            //printf("\tindex:%i i:%i j:%i val:%f\n", index, i, j, maxVal);
            if(saveIndices) {
                float* indices = indexOffset + targetOffset;
                indices[bX * outputRows + bY] = bestOffset;
            }
            target[bX * outputRows + bY] = maxVal;
        }
	}

	__global__ void TensorReverseMaxPool(
        uint size, 
        const float* __restrict a,
        const float* __restrict indices,
        float* __restrict b, 
        uint rows,
        uint columns,
        uint depth,
        uint count,
        uint outputRows,
        uint outputColumns,
        uint filterWidth,
        uint filterHeight,
        uint xStride,
		uint yStride
    ) {
        for (uint index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            uint i = index % rows;
            uint index2 = index / rows;

            uint j = index2 % columns;
            uint index3 = index2 / columns;

            uint k = index3 % depth;
            uint z = index3 / depth;

            uint sourceOffset = (z * rows * columns * depth) + (k * rows * columns);
            const float* source = a + sourceOffset;
            const float* indexPtr = indices + sourceOffset;
            float* target = b + (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            uint sourceIndex = j * rows + i;
            float val = source[sourceIndex];
            uint offset = indexPtr[sourceIndex];

            if(offset < 0)
                offset = 0;

            uint targetX = j * xStride + (offset / filterHeight);
            uint targetY = i * yStride + (offset % filterHeight);

            /*printf("index:%i s:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) val:%f offset:%i tx:%i ty:%i\n", 
                index, xStride, yStride
                i, outputRows,
                j, outputColumns, 
                k, depth, 
                z, count,
                val, offset,
                targetX, targetY
            );*/

            target[targetX * outputRows + targetY] = val;
        }
	}

    __global__ void CalculateDistances(
        const float** __restrict a,
        const float** __restrict b,
        float* __restrict c,
        uint rows,
        uint columns,
        uint size,
        uint distanceMetric
    ) {
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                for (uint k = blockDim.z * blockIdx.z + threadIdx.z; k < rows; k += blockDim.z * gridDim.z) {
                    float aVal = a[j][i];
                    float bVal = b[k][i];
                    float output = 0;

                    if(distanceMetric == 0) { // euclidean
                        float diff = aVal - bVal;
                        output = diff * diff;
                    }else if(distanceMetric == 1) { // cosine
                        output = aVal * bVal;
                    }else if(distanceMetric == 2) { // manhattan
                        output = abs(aVal - bVal);
                    }
                    float* outputPtr = c + (j * rows + k);
                    atomicAdd(outputPtr, output);
                }
            }
        }
	}

	__global__ void MultiCosineDistance(
		const float** __restrict a, 
		const float** __restrict b, 
		float* __restrict aa, 
		float* __restrict ab, 
		float* __restrict bb, 
		uint rows,
        uint columns,
        uint size
	) {
        for (uint i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (uint j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                for (uint k = blockDim.z * blockIdx.z + threadIdx.z; k < rows; k += blockDim.z * gridDim.z) {
					float aVal = a[j][i];
					float bVal = b[k][i];
					uint offset = j * rows + k;
					atomicAdd(aa + offset, aVal * aVal);
					atomicAdd(ab + offset, aVal * bVal);
					atomicAdd(bb + offset, bVal * bVal);
				}
            }
        }
	}
}